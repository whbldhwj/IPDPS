#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <string.h>
#include <sys/time.h>
#include <ctime>
#include "params.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern __global__ void calMatrix(
    int refNum,
    int p_w_match, int p_w_mismatch, int p_w_open, int p_w_extend,
    char *ref, char *alt,
    short *refLen, short *altLen, short *strategy,
    btrack_t *btrack,
    int *sw_lastrow,
    int *sw_lastcol,
    int *best_gap_v,
    char *gap_size_v
);

extern __global__ void calCigar(
    int refNum,
    btrack_t *btrack,
    int *sw_lastrow,
    int *sw_lastcol,
    short *refLen, short *altLen, short *strategy,
    short *state, short *segmentLen, short *offset, short *num
);

float sw_wrap(
    int refNum,
    char *ref, char *alt,
    short *refLen, short *altLen,
    short *strategy,
    short *state, short *segmentLen, short *offset, short *num
){
//    int *sw_lastrow = (int*)malloc(sizeof(int)*PADDED_ALT_LEN*refNum);
//    int *sw_lastcol = (int*)malloc(sizeof(int)*PADDED_REF_LEN*refNum);

//    btrack_t *btrack = (btrack_t*)malloc(sizeof(btrack_t)*PADDED_REF_LEN*PADDED_ALT_LEN*refNum);
    int *best_gap_v = (int*)malloc(sizeof(int)*PADDED_ALT_LEN*refNum);
    for(int i=0; i<refNum; i++){
        for(int j=0; j<PADDED_ALT_LEN; j++){
            best_gap_v[i*PADDED_ALT_LEN + j] = lowInitValue;
        }
    }

//    short refMaxLen = 0;
//    short altMaxLen = 0;
//    for (int i=0; i<refNum; i++){
//        if (refLen[i] > refMaxLen)
//            refMaxLen = refLen[i];
//        if (altLen[i] > altMaxLen)
//            altMaxLen = altLen[i];
//    }
    int *sw_lastrow_cuda;
    int *sw_lastcol_cuda;
    btrack_t *btrack_cuda;
    char *ref_cuda;
    char *alt_cuda;
    short *refLen_cuda;
    short *altLen_cuda;
    short *strategy_cuda;

    int *best_gap_v_cuda;
    char *gap_size_v_cuda;
    
    hipMalloc((void**)& sw_lastrow_cuda, PADDED_ALT_LEN*refNum*sizeof(int));
    hipMalloc((void**)& sw_lastcol_cuda, PADDED_REF_LEN*refNum*sizeof(int));
    hipMalloc((void**)& btrack_cuda, PADDED_REF_LEN*PADDED_ALT_LEN*refNum*sizeof(btrack_t));
    hipMalloc((void**)& ref_cuda, MAX_REF_LEN*refNum*sizeof(char));
    hipMalloc((void**)& alt_cuda, MAX_ALT_LEN*refNum*sizeof(char));
    hipMalloc((void**)& refLen_cuda, refNum*sizeof(short));
    hipMalloc((void**)& altLen_cuda, refNum*sizeof(short));
    hipMalloc((void**)& strategy_cuda, refNum*sizeof(short));

    hipMalloc((void**)& best_gap_v_cuda, refNum*PADDED_ALT_LEN*sizeof(int));
    hipMalloc((void**)& gap_size_v_cuda, refNum*PADDED_ALT_LEN*sizeof(char));

    hipEvent_t start, stop;
    float elapsedTime1;
    float elapsedTime2;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMemset(sw_lastrow_cuda, 0, PADDED_ALT_LEN*refNum*sizeof(int));
    hipMemset(sw_lastcol_cuda, 0, PADDED_REF_LEN*refNum*sizeof(int));
    hipMemcpy(ref_cuda, ref, MAX_REF_LEN*refNum*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(alt_cuda, alt, MAX_ALT_LEN*refNum*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(refLen_cuda, refLen, refNum*sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(altLen_cuda, altLen, refNum*sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(strategy_cuda, strategy, refNum*sizeof(short), hipMemcpyHostToDevice);
    
    hipMemcpy(best_gap_v_cuda, best_gap_v, refNum*PADDED_ALT_LEN*sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(refNum, 1);
    dim3 dimBlock(BLOCK_SIZE,1);

//    hipEventRecord(start, 0);

    calMatrix<<<dimGrid, dimBlock>>>(
        refNum,
        200, -150, -260, -11,
        ref_cuda, alt_cuda,
        refLen_cuda, altLen_cuda, strategy_cuda,
        btrack_cuda,
        sw_lastrow_cuda,
        sw_lastcol_cuda,
        best_gap_v_cuda, 
        gap_size_v_cuda
    );
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);

//    hipMemcpy(sw_lastrow, sw_lastrow_cuda, sizeof(int)*PADDED_ALT_LEN*refNum, hipMemcpyDeviceToHost);
//    hipMemcpy(sw_lastcol, sw_lastcol_cuda, sizeof(int)*PADDED_REF_LEN*refNum, hipMemcpyDeviceToHost);
//    hipMemcpy(btrack, btrack_cuda, sizeof(btrack_t)*PADDED_REF_LEN*PADDED_ALT_LEN*refNum, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime1, start, stop);
   
    short *state_cuda, *segmentLen_cuda, *offset_cuda, *num_cuda;
    hipMalloc((void**)& state_cuda, refNum*MAX_STATE_NUM*sizeof(short));
    hipMalloc((void**)& segmentLen_cuda, refNum*MAX_STATE_NUM*sizeof(short));
    hipMalloc((void**)& offset_cuda, refNum*sizeof(short));
    hipMalloc((void**)& num_cuda, refNum*sizeof(short));

    hipEventRecord(start, 0);

//    dim3 dimGrid(refNum, 1);
//    dim3 dimBlock(1, 1);
    dimBlock.x = 1;
    calCigar<<<dimGrid, dimBlock>>>(  
        refNum,
        btrack_cuda,
        sw_lastrow_cuda,
        sw_lastcol_cuda,
        refLen_cuda, altLen_cuda, strategy_cuda,
        state_cuda, segmentLen_cuda, offset_cuda, num_cuda
    );

    hipMemcpy(state, state_cuda, sizeof(short)*MAX_STATE_NUM*refNum, hipMemcpyDeviceToHost);
    hipMemcpy(segmentLen, segmentLen_cuda, sizeof(short)*MAX_STATE_NUM*refNum, hipMemcpyDeviceToHost);
    hipMemcpy(offset, offset_cuda, sizeof(short)*refNum, hipMemcpyDeviceToHost);
    hipMemcpy(num, num_cuda, sizeof(short)*refNum, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
   
    hipEventElapsedTime(&elapsedTime2, start, stop);

    hipFree(sw_lastrow_cuda);
    hipFree(sw_lastcol_cuda);
    hipFree(btrack_cuda);
    hipFree(ref_cuda);
    hipFree(alt_cuda);
    hipFree(refLen_cuda);
    hipFree(altLen_cuda);
    hipFree(strategy_cuda);   

    hipFree(best_gap_v_cuda);
    hipFree(gap_size_v_cuda);

    hipFree(state_cuda);
    hipFree(segmentLen_cuda);
    hipFree(offset_cuda);
    hipFree(num_cuda);

//    return elapsedTime1+elapsedTime2;
    return elapsedTime1;
}
